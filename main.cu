#include "hip/hip_runtime.h"
#include <unistd.h>
#include "kernels.h"
#include "CycleTimer.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <tuple>
#define IDX(i, j, n) ((i) * (n) + (j))

using namespace std;

typedef vector< tuple<int, int, int> > tuple_list;

struct Graph1 {
    int n;
    int m;
    int *capacities;
    tuple_list edges;

};

struct New_graph {
    Graph* graph;
    int k;
    int* removed_set;
};

struct Var_helper {
    int size;
    float sum;
    float sqsum;
};

Graph1* scan() {
    ifstream myfile("input.txt");
    int n, m, a, b, c;
    myfile >> n >> m;
    Graph1* graph = (Graph1 *)malloc(sizeof(Graph1));
    graph->n = n;
    graph->m = m;
    tuple_list edges;
    int *capacities = (int *)calloc((n * n), sizeof(int));
    while(m--){
        myfile >> a >> b >> c;
        capacities[IDX(a,b,n)] = c;
        edges.push_back( tuple<int, int, int>(c, a, b));
    }
    myfile.close();
    graph->capacities = capacities;
    graph->edges = edges;
    return graph;
}

Graph1* scan_format(int no, int id) {
    string filename = "sample_graphs/g_" + to_string(no) + "_" + to_string(id) + ".txt";
    // filename = "small.txt";
    ifstream myfile(filename);
    int i,j, n, m, a, b, c, edge_count=0;
    string s1,s2,s3;
    string line;
    for(i=0;i<7;i++){
        getline(myfile, line);
    }
    myfile>> s1 >> s2 >> n >> m;

    Graph1* graph = (Graph1 *)malloc(sizeof(Graph1));
    graph->n = n;
    tuple_list edges;
    int *capacities = (int *)calloc((n * n), sizeof(int));
    for(i=0;i<n;i++)
        for(j=0;j<n;j++)
            capacities[IDX(i,j,n)] = 0;
    while(m--){
        myfile >> s1 >> a >> b >> c;
        a--;
        b--;
        c++;
        if(capacities[IDX(a,b,n)] == 0){
            capacities[IDX(a,b,n)] = c;
            edges.push_back( tuple<int, int, int>(c, a, b));
            edge_count++;
        }
    }
    myfile.close();
    graph->m = edge_count;
    graph->capacities = capacities;
    graph->edges = edges;
    return graph;
}

int BFS(Graph *g, int *flowMatrix, int *parents, int *pathCapacities, int s, int t) {
    memset(parents, -1, (g->n * sizeof(int)));
    memset(pathCapacities, 0, (g->n * sizeof(int)));
    parents[s] = s;
    pathCapacities[s] = std::numeric_limits<int>::max();
    std::queue<int> bfsQueue;
    bfsQueue.push(s);
    while (!bfsQueue.empty()) {
        int u = bfsQueue.front();
        bfsQueue.pop();
        for (int v = 0; v < g->n; v++) {
            if (u == v) continue;
            int residual = g->capacities[IDX(u, v, g->n)] - flowMatrix[IDX(u, v, g->n)];
            if ((residual > 0) && (parents[v] == -1)) {
                parents[v] = u;
                pathCapacities[v] = std::min(pathCapacities[u], residual);
                if (v != t) {
                    bfsQueue.push(v);
                } else {
                    int result = pathCapacities[t];
                    return result;
                }
            }
        }
    }
    return 0;
}

// Edmonds-Karp algorithm to find max s-t flow
Flow *edKarpSeq(Graph *g, int s, int t) {
    int flow = 0;
    int *flowMatrix = (int *)calloc((g->n * g->n), sizeof(int));
    int *parents = (int *)malloc(g->n * sizeof(int));
    int *pathCapacities = (int *)calloc(g->n, sizeof(int));
    while (true) {
        int tempCapacity = BFS(g, flowMatrix, parents, pathCapacities, s, t);
        if (tempCapacity == 0) {
            break;
        }
        flow += tempCapacity;
        int v = t;
        // backtrack
        while (v != s) {
            int u = parents[v];
            flowMatrix[IDX(u, v, g->n)] += tempCapacity;
            flowMatrix[IDX(v, u, g->n)] -= tempCapacity;
            v = u;
        }
    }
    Flow *result = (Flow *)malloc(sizeof(Flow));
    result->maxFlow = flow;
    result->finalEdgeFlows = flowMatrix;
    free(parents);
    free(pathCapacities);
    return result;
}

set<int> into_node(Graph* g, int v) {
    int i, n = g->n;
    set<int> ret;
    for(i=0;i<n;i++) {
        if((g->capacities)[IDX(i,v,n)] != 0){
            ret.insert(i);
        }
    }
    return ret;
}

set<int> out_node(Graph* g, int v) {
    int i, n = g->n;
    set<int> ret;
    for(i=0;i<n;i++) {
        if((g->capacities)[IDX(v,i,n)] != 0){
            ret.insert(i);
        }
    }
    return ret;
}

set<int> into_node_intersect(Graph* g, int v, set<int> removal_set) {
    int i, n = g->n;
    set<int> ret;
    for(i=0;i<n;i++) {
        if(removal_set.find(i) != removal_set.end()){
            if((g->capacities)[IDX(v,i,n)] != 0){
                ret.insert(i);
            }
        }
    }
    return ret;
}

set<int> out_node_intersect(Graph* g, int v, set<int> removal_set) {
    int i, n = g->n;
    set<int> ret;
    for(i=0;i<n;i++) {
        if(removal_set.find(i) != removal_set.end()){
            if((g->capacities)[IDX(i,v,n)] != 0){
                ret.insert(i);
            }
        }
    }
    return ret;
}
set<int> into_nodes_set(Graph* g, set<int> removal_set) {
    set<int> ret;
    for(auto x : removal_set) {
        set<int> temp = into_node(g, x);
        ret.insert(temp.begin(), temp.end());
    }
    for(auto x : removal_set) {
        if(ret.find(x) != ret.end())
            ret.erase(x);
    }
    return ret;
}

set<int> out_nodes_set(Graph* g, set<int> removal_set) {
    set<int> ret;
    for(auto x : removal_set) {
        set<int> temp = out_node(g, x);
        ret.insert(temp.begin(), temp.end());
    }
    for(auto x : removal_set) {
        if(ret.find(x) != ret.end())
            ret.erase(x);
    }
    return ret;
}

float into_node_intersect_sum(Graph* g, int v, set<int> removal_set){
    int i, n = g->n;
    float ret=0;
    for(i=0;i<n;i++) {
        if(removal_set.find(i) != removal_set.end()){
            int edge = (g->capacities)[IDX(v,i,n)];
            if(edge != 0){
                ret += edge;
            }
        }
    }
    return ret;
}

float out_node_intersect_sum(Graph* g, int v, set<int> removal_set){
    int i, n = g->n;
    float ret=0;
    for(i=0;i<n;i++) {
        if(removal_set.find(i) != removal_set.end()){
            int edge = (g->capacities)[IDX(v,i,n)];
            if(edge != 0){
                ret += edge;
            }
        }
    }
    return ret;
}

float minimum_edge(Graph* g, set<int> nodes){
    int n = g->n;
    float ret = INT_MAX;
    for(auto x:nodes){
        for(auto y:nodes){
            if(x != y){
                float temp = g->capacities[IDX(x, y, n)];
                if(temp != 0)
                    ret = min(temp, ret);
            }   
        }
    }
    return ret;
}

void del_part_graph(Graph* g, set<int> removal_set, int* connectivity) {
    int i,n = g->n;
    std::vector<tuple<int, int, int> > edges;
    set<int> into_nodes = into_nodes_set(g, removal_set) ;
    set<int> out_nodes = out_nodes_set(g, removal_set);
    float minedge = minimum_edge(g, removal_set);
    int k = removal_set.size();
   for(auto x: into_nodes){
        for(auto y: out_nodes){
            set<int> inner_into = into_node_intersect(g, x, removal_set);
            set<int> inner_out = out_node_intersect(g, y, removal_set);
            float replace = minedge;
            int flag = 0;
            for(auto a: inner_into){
                for(auto b: inner_out){
                    if((a == b) || connectivity[IDX(a,b,n)]){
                        (g->capacities)[IDX(x,y,n)] = replace;
                        flag = 1;
                        break;
                    }
                }
                if(flag) {
                    break;
                }
            }
        }
    }

    // Disconnecting the removaal set from the rest graph
    for(auto x:removal_set){
        for(i=0;i<n;i++){
            (g->capacities)[IDX(i,x,n)] = 0;
            (g->capacities)[IDX(x,i,n)] = 0;
        }
    }
}

New_graph* findRemovedSet(Graph1* g, tuple_list edges, set<int> imp, int* connectivity, float MAXVAR) {
    int i,m,n,a,b;
    m = edges.size();
    n = g->n;;
    float c;
    bool debugFlag = false;
    sort(edges.begin(), edges.end());
    if(debugFlag){
        for(i=0;i<m;i++)
            cout<<get<0>(edges[i])<<" "<<get<1>(edges[i])<<" "<<get<2>(edges[i])<<endl;
    }

    Var_helper* var_helper = (Var_helper*)malloc(sizeof(Var_helper));
    var_helper->size = 0;
    var_helper->sum = 0;
    var_helper->sqsum = 0;
    set<int> removal_set;
    set<int> removed_set;
    Graph* g1 = (Graph *)malloc(sizeof(Graph));
    g1->n = g->n;
    g1->capacities = (int *)malloc(n*n*sizeof(int));
    memcpy(g1->capacities, g->capacities, n*n*sizeof(int));
    for(i=0;i<m;i++){
        a = get<1>(edges[i]);
        b = get<2>(edges[i]);
        c = get<0>(edges[i]);
        if((removed_set.find(a) == removed_set.end()) && (removed_set.find(b) == removed_set.end())){
            if((imp.find(a) == imp.end()) && (imp.find(b) == imp.end())) {
                float sum = var_helper->sum;
                float sqsum = var_helper->sqsum;
                int k = var_helper->size;
                k++;
                sum += c;
                sqsum += (c*c);
                float var = (sqsum/k) - (sum/k)*(sum/k);
                if(debugFlag){
                    cout<<i<<" :"<<m<<": "<<c<<" --- "<<var<<" ----> "<<k<<endl;
                }
                if(var < MAXVAR){
                    var_helper->sum = sum;
                    var_helper->sqsum = sqsum;
                    var_helper->size = k;
                    removal_set.insert(a);
                    removal_set.insert(b);
                    if(debugFlag){
                        cout<<"inserted "<<a<<" "<<b<<endl;
                    }
                   
                }else {
                    if(removal_set.size() > 2) { 
                        removed_set.insert(removal_set.begin(), removal_set.end());
                        if(debugFlag){
                            for(auto x:removal_set){
                                cout<< x ;
                                cout<< " ";
                            }
                            cout<<endl;
                        }
                        del_part_graph(g1, removal_set, connectivity);
                    }
                    removal_set.clear();
                    i--;
                    var_helper->sum = 0;
                    var_helper->sqsum = 0;
                    var_helper->size = 0;
                }
                
            }
        }
    }
    removed_set.insert(removal_set.begin(), removal_set.end());
    if(debugFlag){
        for(auto x:removal_set){
            cout<< x ;
            cout<< " ";
        }
        cout<<endl;
    }
    del_part_graph(g1, removal_set, connectivity);
    removal_set.clear();
    int k = removed_set.size();
    New_graph* ret = (New_graph*)malloc(sizeof(New_graph));
    ret->graph = g1;
    ret->removed_set = (int*) malloc(k*sizeof(int));
    ret->k = k;
    i=0;
    for(auto x:removed_set){
        (ret->removed_set)[i++] = x;
    }
    return ret;
}

Graph* converted_graph(Graph* g, vector<int> consi_vert){
    Graph* ret = (Graph*)malloc(sizeof(Graph));
    int k = consi_vert.size();
    int n = g->n;
    ret->n = k;
    ret->capacities = (int*)malloc(k*k*sizeof(int));
    int i = 0,j=0;
    for(i=0;i<k;i++){
        for(j=0;j<k;j++){
            (ret->capacities)[IDX(i,j,k)] = (g->capacities)[IDX(consi_vert[i],consi_vert[j],n)];
        }   
    }
    return ret;
}

int main(int argc, char** argv) {
    if(argc != 4){
        cout<< "Command Format"<<endl;
        cout<<"./main N id MAXVAR"<<endl;
        cout<< "N -> No. of vertices(100, 1000)"<<endl;
        cout<< "id -> Index among graphs(0 to 49)"<<endl;
        return;
    }
    srand (time(NULL));
    int no = atoi(argv[1]);
    int id = atoi(argv[2]);
    float MAXVAR = stof(argv[3]);
	Graph1 *graph = scan_format(no,id);
	int i,n = graph->n;
    double start, finalTime, origTime, changeTime, preTime;
    int origFlow,changeFlow;
    
    //********** Get connectivity of vertices here****************
	int *W, *con;
	con = (int*)malloc(n*n*sizeof(int));
	hipMalloc(&W, n*n*sizeof(int));
	hipMemcpy(W, graph->capacities, n*n*sizeof(int), hipMemcpyHostToDevice);
    start = CycleTimer::currentSeconds();
    for(int k=0; k<n; k++) {
        parallel_floyd_warshall<<<1, n*n>>>(n, k, W);
        hipDeviceSynchronize();
    }
    finalTime = CycleTimer::currentSeconds() - start;
    // printf("connectivity took %f\n", finalTime);
    hipMemcpy(con, W, n*n*sizeof(int), hipMemcpyDeviceToHost);
    // *********** Connectivity part ends here *********************



    int s = rand()%n;
    int t= s;
    while(t == s){
        t = rand()%n;
    }
    set<int> imp;
    imp.insert(s);
    imp.insert(t);

   
    tuple_list edges = graph->edges;
    start = CycleTimer::currentSeconds();
    New_graph* ret = findRemovedSet(graph, edges, imp, con, MAXVAR);
    preTime = CycleTimer::currentSeconds() - start;
    int k = ret->k;
    int* removed_set = ret->removed_set;
    Graph* g1 = ret->graph;

    set<int> consi_vert;
    for(i=0;i<n;i++)
        consi_vert.insert(i);

    for(i=0;i<k;i++)
        consi_vert.erase(removed_set[i]);

    vector<int> v(consi_vert.begin(), consi_vert.end());
    sort(v.begin(),v.end());
    Graph* conv_graph = converted_graph(g1, v);

	Graph *g = (Graph *)malloc(sizeof(Graph));
    g->n = graph->n;
    g->capacities = graph->capacities;
    Flow *result;


    start = CycleTimer::currentSeconds();
    result = edKarpSeq(g, s, t);
    origTime = CycleTimer::currentSeconds() - start;
    origFlow = result->maxFlow;
    // ***** Application Ends here **********************************
    int news = distance(consi_vert.begin(), consi_vert.find(s));
    int newt = distance(consi_vert.begin(), consi_vert.find(t));
    start = CycleTimer::currentSeconds();
    result = edKarpSeq(conv_graph, news, newt);
    changeTime = CycleTimer::currentSeconds() - start;
    changeFlow = result->maxFlow;
    printf("%d %d %f %f %d %f %f %d \n", no,id, MAXVAR, origTime, origFlow, preTime+finalTime, changeTime, changeFlow); 
    consi_vert.clear();
    
    return 0;
}